#include "hip/hip_runtime.h"
/** 
 This sample is designed to test CUDA-OpenGL interoperability performance in a system.
- Requires CUDA version 5.0 or greater.
- It creates a GLUT window and tests mapping/unmapping of a texture of a given size
  using GL/CUDA interoperability on every CUDA device available in the system.
- CUDA is the producer and OpenGL is the consumer in this benchmark. This model is implemented by having
  the interoperability mapping flags set to cudaGraphicsMapFlagsWriteDiscard.
- The application computes "speedup" for each CUDA/GL context configuration possible in a given system.
  The speedup is the frame rate compared to pure workload(no interoperability) framerate 
- The application also auto-adjusts the time it spends in GL work and CUDA kernels
  so that it simulates a specified workload (default is 60 FPS).  This is because
  the number of iterations is sensitive to compiler changes, clocks, GPU type, etc.
 **/


#if defined(_WIN32)
#include <windows.h>
#include <stdint.h>
#define strcasecmp stricmp

#else /* _UNIX */
#include <sys/time.h>
#include <stdint.h>
#include <stdbool.h>
#include <stddef.h>
#include <assert.h>
#include <string.h>

#endif /* _UNIX */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <GL/glew.h>
#include <GL/glut.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

// Important performance knobs to tune

// Texture dimentions set by the user
// - in multi-GPU interop, the whole texture needs to be copied between 
//   the two GPUs on every frame, so this size will modify that amount
//   of data copied
// - each pixel in the texture is 4 bytes (so 32 means 32*32*4 = 4k bytes)
int texWidth = 1920;
int texHeight = 1080;

// The time to spend doing CUDA work set by the user
// - this allows you to simulate different amounts of CUDA work being done
//   per frame (the default is fully-loaded at ~60 fps)
// - the application will spend this many msec running CUDA work
// - this is used to compute knobItersCUDA
float targetMsecCUDA = 17.f;

// The time to spend doing OpenGL work set by the user
// - this allows you to simulate different amounts of OpenGL rendering being done
//   per frame (the default is fully-loaded at ~60 fps)
// - the application will spend this many msec running OpenGL rendering
// - this is used to compute knobItersGL
float targetMsecGL = 17.f;


#define GL_CHECK_ERROR() \
    do { \
        GLenum err; \
        err = glGetError(); \
        if(err != GL_NO_ERROR) \
        { \
            fprintf(stderr, "%s:%d GL error: %s\n", __FILE__, __LINE__, gluErrorString(err)); \
            exit(1); \
        } \
    } while (0)


#define CUDA_CHECK_ERROR() \
    do { \
        hipError_t err = hipGetLastError(); \
        if (hipSuccess != err) { \
            fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, __LINE__, hipGetErrorString(err) ); \
            exit(1); \
        } \
    } while(0) 

unsigned char *cudaBuffer;
hipGraphicsResource_t interopTexHandle;
GLuint interopTex;
GLuint vshader;
GLuint fshader;
GLuint program;
GLint programTexLoc;
GLint programItersLoc;
GLint programValueLoc;

GLuint glTimerQuery;

int knobItersCUDA = 64*1024;
int knobItersGL = 64*1024;

hipStream_t cudaStream = NULL;
struct hipArray *interopTexArr = NULL;
hipEvent_t 	cudaTaskStart;
hipEvent_t 	cudaTaskEnd;

int cudaDevice = 0;

#if defined(_WIN32)

inline bool QueryPerformanceFrequency(int64_t *frequency)
{
	LARGE_INTEGER f;
	bool ret = QueryPerformanceFrequency(&f);
	*frequency = f.QuadPart;
	return ret;
}

inline bool QueryPerformanceCounter(int64_t *performance_count)
{
	LARGE_INTEGER p;
	bool ret = QueryPerformanceCounter(&p);
	*performance_count = p.QuadPart;
	return ret;
}

#else  /* _UNIX */
/* Helpful conversion constants. */
static const unsigned usec_per_sec = 1000000;


/* These functions are written to match the win32
   signatures and behavior as closely as possible.
*/
bool QueryPerformanceFrequency(int64_t *frequency)
{
    /* Sanity check. */
    assert(frequency != NULL);

    /* gettimeofday reports to microsecond accuracy. */
    *frequency = usec_per_sec;

    return true;
}


bool QueryPerformanceCounter(int64_t *performance_count)
{
    struct timeval time;

    /* Sanity check. */
    assert(performance_count != NULL);

    /* Grab the current time. */
    gettimeofday(&time, NULL);
    *performance_count = time.tv_usec + /* Microseconds. */
                         time.tv_sec * usec_per_sec; /* Seconds. */

    return true;
}


#endif /* _UNIX */
//
// Return nanosecond clock value.
//
int64_t GetNanoClock()
{
	int64_t now;
	static int64_t frequency;
	static int gotfrequency = 0;
	int64_t seconds, nsec;

	QueryPerformanceCounter(&now);
	if (gotfrequency == 0) {
		QueryPerformanceFrequency(&frequency);
		gotfrequency = 1;
	}

	seconds = now / frequency;
	nsec = (1000000000 * (now - (seconds * frequency))) / frequency;
    
	return seconds * 1000000000 + nsec;
}


// Simple CUDA kernel designed to waste a whole bunch of cycles also doing nothing
__global__ void K(unsigned char *cudaBuffer, unsigned char value, int iters)
{
    unsigned char *pixel = cudaBuffer + 4*threadIdx.x;
    pixel[0] = value;
    pixel[1] = value;
    pixel[2] = value;
    pixel[3] = value;

    for (int i = 0; i < iters; ++i) {
        pixel[0] = pixel[0] * pixel[1] + i*i;
    }
    pixel[0] &= 0xF0;
    pixel[0] |= value;
}


void buildProgram(void)
{
    const char *vtext = "\
        void main() \n\
        { \n\
            gl_FrontColor = gl_Color; \n\
            gl_Position = ftransform(); \n\
        }";
    const char *ftext = "\
        uniform sampler2D tex; \n\
        uniform int iters; \n\
        uniform float value; \n\
        void main() \n\
        { \n\
            float x = value; \n\
            for (int i = 0; i < iters; ++i) \n\
             { \n\
                x = sqrt(x) + 0.001; \n\
            } \n\
            gl_FragColor = x*gl_Color*texture2D(tex,gl_TexCoord[0].st); \n\
        }";
    vshader = glCreateShader(GL_VERTEX_SHADER);
    fshader = glCreateShader(GL_FRAGMENT_SHADER); 
    glShaderSource(vshader, 1, &vtext, NULL);
    glShaderSource(fshader, 1, &ftext, NULL);
    glCompileShader(vshader);
    glCompileShader(fshader);
    program = glCreateProgram();
    glAttachShader(program, vshader);
    glAttachShader(program, fshader);
    glLinkProgram(program);
    programTexLoc   = glGetUniformLocation(program, "tex");
    programItersLoc = glGetUniformLocation(program, "iters");
    programValueLoc = glGetUniformLocation(program, "value");
    GL_CHECK_ERROR();
}


float tuneLinearKnob(int *knob, float (*eval)(void), float target)
{
    int steps = 0;
	float value;
    while (1) {
        value = eval();
        int knobValue = *knob;
        printf("%1.2f with %d iterations...", value, knobValue);

        *knob = (int)( knobValue*(target/value) );
        if (*knob <= 0) {
            *knob = 1;
        }

        steps += 1;
        if (steps == 2) {
            break;
        }
    }
	return value;
}


// Execute a long-running kernel in CUDA, then copy data to the interop hipArray_t
void doWorkInCUDA(void)
{
    // Run a kernel that wastes a lot of time
    static char value = 0;
    value += 1;
    K<<<1,1,0,cudaStream>>>(cudaBuffer, value, knobItersCUDA);
    CUDA_CHECK_ERROR();

}



// Draw using the interop GL texture
void doWorkInGL(void)
{
    glBindFramebufferEXT(GL_FRAMEBUFFER_EXT, 0);
    glViewport(0, 0, 512, 512);
    glClearColor(0, 0, 0, 1);
    glClear(GL_COLOR_BUFFER_BIT);

    // Make a really expensive draw call which uses the interop texture
    // - we make the call expensive by doing lots of iterations in a
    //   loop in the pixel shader
    // - we use a really small viewport so that the number of processors
    //   in the GPU doesn't impact performance (rather, only clocks matter,
    //   and clocks don't change as rapidly across generations)
    {
        glViewport(0, 0, 4, 4);

        glEnable(GL_TEXTURE_2D);
        glBindTexture(GL_TEXTURE_2D, interopTex);
        glUseProgram(program);
        glUniform1i(programTexLoc,   0);
        glUniform1i(programItersLoc, knobItersGL);
        glUniform1f(programValueLoc, 1.001);        

        glBegin(GL_QUADS);
        glColor3f(1, 1, 1); 
        glTexCoord2f(1.0f, 1.0f); glVertex3f( 0.8,  0.8, 0);
        glTexCoord2f(0.0f, 1.0f); glVertex3f(-0.8,  0.8, 0);
        glTexCoord2f(0.0f, 0.0f); glVertex3f(-0.8, -0.8, 0);
        glTexCoord2f(1.0f, 0.0f); glVertex3f( 0.8, -0.8, 0);
        glEnd();

        glDisable(GL_TEXTURE_2D);
        glBindTexture(GL_TEXTURE_2D, 0);
        glUseProgram(0);
    }

	
    // Draw a triangle
    glViewport(0, 0, 512, 512);
    {
        glBegin(GL_TRIANGLES);
        glTexCoord2f(0.0f, 0.0f); glColor3f(0, 0, 1); glVertex3f(-0.6, -0.6, 0);
        glTexCoord2f(0.0f, 1.0f); glColor3f(0, 1, 0); glVertex3f(-0.6,  0.6, 0);
        glTexCoord2f(1.0f, 1.0f); glColor3f(1, 0, 0); glVertex3f( 0.6, -0.6, 0);
        glEnd();
    }
}

int timingIterations = 100;

// Benchmark just the CUDA kernel alone
float benchmarkCUDA(void)
{

    float msecCUDA = 0.f;

    hipGraphicsMapResources(1, &interopTexHandle, cudaStream);
	CUDA_CHECK_ERROR();	
	hipGraphicsSubResourceGetMappedArray(&interopTexArr, interopTexHandle, 0, 0);
	CUDA_CHECK_ERROR();	
	//GPU based timing is commented out but can be used for timing as well.
	
	hipStreamSynchronize(cudaStream);
	int64_t start, end;
	start = GetNanoClock();
	
	//hipEventRecord(cudaTaskStart,cudaStream);
	//CUDA_CHECK_ERROR();
	for (int i = 0; i < timingIterations; ++i) {

		doWorkInCUDA();
    }
	
	
	
	//hipEventRecord(cudaTaskEnd,cudaStream);
	//CUDA_CHECK_ERROR();
    //hipEventSynchronize(cudaTaskEnd);
	//CUDA_CHECK_ERROR(); 
    //hipEventElapsedTime(&msecCUDA, cudaTaskStart, cudaTaskEnd);
	//CUDA_CHECK_ERROR();
	//msecCUDA /= timingIterations;
	
	hipStreamSynchronize(cudaStream);
	end = GetNanoClock();
	msecCUDA = end - start;
	msecCUDA /= 1000000.0f;
	msecCUDA /= timingIterations;

	hipGraphicsUnmapResources(1, &interopTexHandle, cudaStream);	
	CUDA_CHECK_ERROR();  
	

	return msecCUDA;
}


// Benchmark just the GL rendering alone
float benchmarkGL()
{
    float msecGL = 0.f;
	//GPU based timing is commented out but can be used for timing as well.
	 glFinish();   
	int64_t start, end;
	start = GetNanoClock();
	
	//glBeginQuery(GL_TIME_ELAPSED_EXT, glTimerQuery);	
    for (int i = 0; i < timingIterations; ++i) {
        doWorkInGL();
    }
	//glEndQuery(GL_TIME_ELAPSED_EXT);
	//GL_CHECK_ERROR(); 
	//glFinish: this is necessary call to make when timing to make
	//sure that all the preceding OpenGL commands are done before we get the time
    glFinish();    

	//GLuint64EXT timeElapsed = 0;
	//glGetQueryObjectui64vEXT(glTimerQuery, GL_QUERY_RESULT, &timeElapsed);
	//GL_CHECK_ERROR(); 
	//msecGL = timeElapsed*0.000001;	
	//msecGL /= timingIterations;

	end = GetNanoClock();
	msecGL = end - start;
	msecGL /= 1000000.0f;
	msecGL /= timingIterations;
	
    return msecGL;
}



// Benchmark CUDA and GL, with data shared through interop
float benchmarkInterop()
{
	//Must use CPU based timing here since there is a mix of APIs with interdependant calls that
	//are possibly not CPU asynchronous.
    float msecInterop = 0.f;
	//hipStreamSynchronize: this is necessary call to make when doing CPU based timings to make
	//sure that all the preceding CUDA commands are done before we start timing
	hipStreamSynchronize(cudaStream);		
	CUDA_CHECK_ERROR();  
	//glFinish: this is necessary call to make when doing CPU based timings to make
	//sure that all the preceding OpenGL commands are done before we start timing
	glFinish();
	
	int64_t start, end;
	//start timing
	start = GetNanoClock();
    for (int i = 0; i < timingIterations; ++i) {
        hipGraphicsMapResources(1, &interopTexHandle, cudaStream);
		CUDA_CHECK_ERROR();  
        hipGraphicsSubResourceGetMappedArray(&interopTexArr, interopTexHandle, 0, 0);
		CUDA_CHECK_ERROR();  
        doWorkInCUDA();
        hipGraphicsUnmapResources(1, &interopTexHandle, cudaStream);
		CUDA_CHECK_ERROR();  
        doWorkInGL();

    }
	
	
	
	//glFinish: this is necessary call to make when doing CPU based timings to make
	//sure that all the preceding OpenGL commands are done before we finish timing
	glFinish();
	//hipStreamSynchronize: this is necessary call to make when doing CPU based timings to make
	//sure that all the preceding CUDA commands are done before we finish timing
	hipStreamSynchronize(cudaStream);
	CUDA_CHECK_ERROR();  
	//end timing
	end = GetNanoClock();
	
	msecInterop = end - start;
	msecInterop /= 1000000.0f;
	msecInterop /= timingIterations;
	return msecInterop;
}


void benchmark(int cudaDevice, int glDevices[], int glDevicesCount)
{
    printf("\n");
    printf("CUDA is using device %d\n", cudaDevice);
	bool multiGPU = false;
	for(int i = 0; i < glDevicesCount; i++)
	{
		if(glDevices[i] == cudaDevice)
		{
			multiGPU = false;
			printf("CUDA and OpenGL contexts share the same device\n");
		}
		else
		{
			multiGPU = true;
			printf("CUDA and OpenGL contexts reside on different devices\n");
		}
	}
	printf("  Benchmarking using a %dx%d texture:\n",  texWidth, texHeight);
    printf("  Tuning knobs to make CUDA section take ~%1.2f msec... ", targetMsecCUDA);
    tuneLinearKnob(&knobItersCUDA, benchmarkCUDA, targetMsecCUDA);
    printf("\n");

    printf("  Benchmarking CUDA part of the program\n");
    float msecCUDA = benchmarkCUDA();
    printf("    %f msec/frame\n", msecCUDA);

    printf("  Benchmarking GL part of the program\n");
    float msecGL = benchmarkGL();
    printf("    %f msec/frame\n", msecGL);

    printf("  Benchmarking CUDA and GL combined, using interop\n");
    float msecInterop = benchmarkInterop();
    printf("    %f msec/frame\n", msecInterop);  

	float speedup = (msecCUDA+msecGL)/msecInterop;
	printf("    Speedup(versus pure workload): %.2fx\n", speedup);		
		
	if(multiGPU)
	{
		float maxWorkloadTime = max(msecCUDA,msecGL);
		float overhead = msecInterop - maxWorkloadTime;
		printf("    Overhead: %.2f msec/frame\n",overhead);
		printf("    Overhead: %.2f%%\n", overhead*100.0/msecInterop);
	}
	else
	{
		float overhead = msecInterop - msecCUDA - msecGL;
		printf("    Overhead: %.2f msec/frame\n", overhead);
		printf("    Overhead: %.2f%%\n", overhead*100.0/msecInterop);
	}
}



int ParseCommandLine(int count, char **argv)
{	
	// Parse command line arguments
	for(int i = 0; i < count;)
	{
		const char *szBuffer = argv[i++];
		
		if (!strcasecmp("-help", szBuffer)) {
			printf("\n");
			printf("MultiGPU CUDA/OpenGL Interoperability Benchmark - Help\n\n");			
			printf("Program parameters:\n");
			printf("\t-width #\t\t\t: width of the interop texture. Default 1920.\n");
			printf("\t-height #\t\t\t: height of the interop texture. Default 1080.\n");
			printf("\t-targetMsecCUDA #\t\t: The time to spend doing CUDA work in msec(can be fractions). Default 17msec.\n");
			printf("\t-targetMsecGL #\t\t\t: The time to spend doing OpenGL work in msec(can be fractions). Default 17msec.\n");			
			return 0;
		}
		if (!strcasecmp("-width", szBuffer)) {
			if(i == count)
				return -1;
			szBuffer = argv[i++];		
			texWidth = atoi(szBuffer);
		}else if (!strcasecmp("-height", szBuffer)) {
			if(i == count)
				return -1;
			szBuffer = argv[i++];	
			texHeight = atoi(szBuffer);
		}else if(!strcasecmp("-targetMsecCUDA", szBuffer)) {
			if(i == count)
				return -1;
			szBuffer = argv[i++];	
			targetMsecCUDA = atof(szBuffer);			
		}else if(!strcasecmp("-targetMsecGL", szBuffer)) {
			if(i == count)
				return -1;
			szBuffer = argv[i++];	
			targetMsecGL = atof(szBuffer);			
		}
		else
			return -1;
	}

	return 1;
}




int main(int argc, char *argv[])
{
    int ret = ParseCommandLine(argc-1,&argv[1]);
	if(ret < 0)
	{
		printf("Incorrect parameters\n");
		return -1;
	}
	else if(ret == 0)
	{
		return 0;
	}
	else
    {
        unsigned int glDeviceCount = 16;
        int glDevices[16];
        int window;

        glutInit(&argc, argv);
        glutInitDisplayMode(GLUT_RGBA|GLUT_DOUBLE);		
        window = glutCreateWindow("OpenGL Window");
        glutInitWindowSize(512, 512);
        glutInitWindowPosition(0, 0);
		glutHideWindow();
        glewInit();		
        // Create a shader which is super-inefficient, which we'll use to simulate a complex draw callcalls
        buildProgram();

		//glGenQueries(1,&glTimerQuery);

        // Print all devices that are spanned by the current GL context.
        hipGLGetDevices(&glDeviceCount, glDevices, glDeviceCount, hipGLDeviceListAll);
        CUDA_CHECK_ERROR();
        printf("OpenGL is using CUDA device(s): ");
        for (unsigned int i = 0; i < glDeviceCount; ++i) {
            printf("%s%d", i == 0 ? "" : ", ", glDevices[i]);
        }
		
        printf("\n");
		
	    // Allocate the texture we'll use for interop
        glGenTextures(1, &interopTex);
        glBindTexture(GL_TEXTURE_2D, interopTex);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, texWidth, texHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAX_LEVEL, 0);
        glBindTexture(GL_TEXTURE_2D, 0);            
		GL_CHECK_ERROR();
	
		printf("  Tuning knobs to make GL section take ~%1.3f msec... ", targetMsecGL);
		tuneLinearKnob(&knobItersGL, benchmarkGL, targetMsecGL);
		printf("\n");					

		// Benchmark with CUDA
        int cudaDeviceCount = 0;
        hipGetDeviceCount(&cudaDeviceCount);

        CUDA_CHECK_ERROR();
        for (cudaDevice = 0; cudaDevice < cudaDeviceCount; ++cudaDevice) {
      
            hipSetDevice(cudaDevice);
            CUDA_CHECK_ERROR();
			//hipEventCreate(&cudaTaskStart);
			//CUDA_CHECK_ERROR();
			//hipEventCreate(&cudaTaskEnd);
			//CUDA_CHECK_ERROR();
			hipFree(0);
            CUDA_CHECK_ERROR();
            hipGraphicsGLRegisterImage(&interopTexHandle, interopTex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
            CUDA_CHECK_ERROR();
            cudaGraphicsResourceSetMapFlags(interopTexHandle, cudaGraphicsMapFlagsWriteDiscard);
            CUDA_CHECK_ERROR();
            hipStreamCreate(&cudaStream);
            CUDA_CHECK_ERROR();
            hipMalloc(&cudaBuffer, texWidth*texHeight*4);
            CUDA_CHECK_ERROR();

            benchmark(cudaDevice, glDevices, glDeviceCount);
			
			hipStreamDestroy(cudaStream);
			CUDA_CHECK_ERROR();
			//hipEventDestroy(cudaTaskStart);  
			//CUDA_CHECK_ERROR();
			//hipEventDestroy(cudaTaskEnd);  
			//CUDA_CHECK_ERROR();
			hipGraphicsUnregisterResource(interopTexHandle);
            CUDA_CHECK_ERROR();
            hipDeviceReset();
            CUDA_CHECK_ERROR();
			
        }
       // Delete the GL texture
        glDeleteTextures(1, &interopTex);
        GL_CHECK_ERROR();

		//glDeleteQueries(1,&glTimerQuery);
        glutDestroyWindow(window);
    }

    return 0;
}

